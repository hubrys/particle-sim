#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.h"
#include <iostream>

typedef struct
{
    float2* positions;
    float2* velocities;
    int count;
    float deltaTime;
    float particleMass;
    float2 mousePos;
    float mouseMass;
    float friction;
    float2 bounds;
} SOAKernelArgs;

__global__ void tickMouseOnly(SOAKernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float2 position = args.positions[index];
    float2 velocity = add(args.velocities[index],
        scale(
        calculateForceMouse(position.x, position.y, args.particleMass, args.mousePos.x, args.mousePos.y, args.mouseMass),
        args.deltaTime
        ));


    position.x += velocity.x * args.deltaTime;
    position.y += velocity.y * args.deltaTime;

    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
    }


    //if (args.bounds.x - abs(position.x) < 0 &&
    //    position.x * velocity.x > 0)
    //{
    //    //velocity.x = -velocity.x;
    //    position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    //}

    //if (args.bounds.y - abs(position.y) < 0 &&
    //    position.y * velocity.y > 0)
    //{
    //    //velocity.y = -velocity.y;
    //    position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    //}

    velocity = scale(velocity, 1 - args.friction * args.deltaTime);

    args.positions[index] = position;
    args.velocities[index] = velocity;
}

__global__ void tickNBody(SOAKernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float2 position = args.positions[index];
    float2 force = calculateForceMouse(position.x, position.y, args.particleMass,
                                  args.mousePos.x, args.mousePos.y, args.mouseMass);

    //// Calculate particle forces
    //__shared__ float2 positions[BLOCK_SIZE];
    //for (int particleI = 0; particleI < args.count; particleI += BLOCK_SIZE)
    //{
    //    positions[threadIdx.x] = args.positions[particleI + threadIdx.x];
    //    __syncthreads();

    //    for (int subI = 0; subI < BLOCK_SIZE; subI++)
    //    {
    //        force = add(force, calculateForce(
    //            position.x, position.y, args.particleMass,
    //            positions[subI].x, positions[subI].y, args.particleMass)
    //            );
    //    }
    //}

    for (int particleI = 0; particleI < args.count; particleI ++)
    {
    if (particleI != index)
    {
        force = add(force, calculateForce(
                        position.x, position.y, args.particleMass,
                        args.positions[particleI].x, args.positions[particleI].y, args.particleMass)
                        );
    }
    }


    // Calc resulting velocity
    float2 velocity = add(args.velocities[index],
                        scale(force, args.deltaTime)); // velocty += force * deltaTime
    position = add(position, scale(velocity, args.deltaTime));

    /*if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
        {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
        }

        if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
        {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
        }
        */
    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        //velocity.x = -velocity.x;
        position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        //velocity.y = -velocity.y;
        position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    }

    velocity = scale(velocity, 1 - args.friction * args.deltaTime);

    args.positions[index] = position;
    args.velocities[index] = velocity;
}


void launchSOACudaTick(float deltaTime, float particleMass, float mouseX, float mouseY, float mouseMass,
struct hipGraphicsResource* positionsResource, struct hipGraphicsResource* velocitiesResource, bool mouseOnly, int particleCount, float2 bounds, float friction)
{
    size_t numBytes;
    SOAKernelArgs args;
    float2* positions;
    float2* velocities;
    hipError_t result = hipGraphicsMapResources(1, &positionsResource, 0);
    if (result != hipSuccess)
    {
        printf("failed mapping resource\n");
        exit(0);
    }

    result = hipGraphicsResourceGetMappedPointer((void**) &positions, &numBytes, positionsResource);
    if (result != hipSuccess)
    {
        printf("failed grabbing resource\n");
        exit(0);
    }

    result = hipGraphicsMapResources(1, &velocitiesResource, 0);
    if (result != hipSuccess)
    {
        printf("failed mapping v resource\n");
        exit(0);
    }

    result = hipGraphicsResourceGetMappedPointer((void**) &velocities, &numBytes, velocitiesResource);
    if (result != hipSuccess)
    {
        printf("failed grabbing v resource\n");
        exit(0);
    }

    args.positions = positions;
    args.velocities = velocities;
    args.count = particleCount;
    args.deltaTime = deltaTime;
    args.particleMass = particleMass;
    args.mousePos = make_float2(mouseX, mouseY);
    args.mouseMass = mouseMass;
    args.friction = friction;
    args.bounds = bounds;

    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(particleCount / BLOCK_SIZE, 1, 1);

    if (mouseOnly)
    {
        tickMouseOnly<<<gridDim, blockDim>>>(args);
    }
    else
    {
        tickNBody<<<gridDim, blockDim>>>(args);
    }

    hipDeviceSynchronize();

    result = hipGraphicsUnmapResources(1, &positionsResource, 0);
    if (result != hipSuccess)
    {
        printf("failed unmapping resource\n");
        exit(0);
    }

    result = hipGraphicsUnmapResources(1, &velocitiesResource, 0);
    if (result != hipSuccess)
    {
        printf("failed unmapping v resource\n");
        exit(0);
    }
}
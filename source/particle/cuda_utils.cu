#include "hip/hip_runtime.h"
#include "cuda_utils.h"

__device__ float2 add(float2 lhs, float2 rhs)
{
    return make_float2(lhs.x + rhs.x, lhs.y + rhs.y);
}

__device__ float length(float2 vec)
{
    return sqrt(vec.x * vec.x + vec.y * vec.y);
}

__device__ float2 subtract(float2 lhs, float2 rhs)
{
    return make_float2(lhs.x - rhs.x, lhs.y - rhs.y);
}

__device__ float2 normalize(float2 vec)
{
    float l = 1 / length(vec);
    return make_float2(vec.x * l, vec.y * l);
}

__device__ float2 scale(float2 vec, float scale)
{
    return make_float2(vec.x * scale, vec.y * scale);
}

__device__ float2 calculateForce(float x1, float y1, float m1, float x2, float y2, float m2)
{
    float2 diff = make_float2(x2 - x1, y2 - y1);
    float distance = length(diff);
    if (distance != 0)
    {
        distance += MIN_CALC_DISTANCE;
        float magnitude = GRAV_CONST * m1 * m2 / (distance * distance);
        return scale(normalize(diff), magnitude);
    }
    else
    {
        return make_float2(0, 0);
    }
}

__device__ float2 calculateForceMouse(float x1, float y1, float m1, float x2, float y2, float m2)
{
    float2 diff = make_float2(x2 - x1, y2 - y1);
    float distance = length(diff);
    if (distance > 0)
    {
        distance += MIN_CALC_DISTANCE;
        float magnitude = GRAV_CONST * m1 * m2 / distance;
        return scale(normalize(diff), magnitude);
    }
    else
    {
        return make_float2(0, 0);
    }
}
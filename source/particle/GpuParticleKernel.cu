#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.h"
#include <iostream>

typedef struct
{
    float4* particles;
    int count;
    float deltaTime;
    float particleMass;
    float2 mousePos;
    float mouseMass;
    float friction;
    float2 bounds;
} KernelArgs;

__global__ void tickMouseOnly(KernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 particle = args.particles[index];

    float2 velocity = scale(
        calculateForceMouse(particle.x, particle.y, args.particleMass, args.mousePos.x, args.mousePos.y, args.mouseMass),
        args.deltaTime
        );

    velocity.x += particle.z;
    velocity.y += particle.w;
    float2 position = make_float2(particle.x, particle.y);

    position.x += velocity.x * args.deltaTime;
    position.y += velocity.y * args.deltaTime;

    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
    }


    //if (args.bounds.x - abs(position.x) < 0 &&
    //    position.x * velocity.x > 0)
    //{
    //    //velocity.x = -velocity.x;
    //    position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    //}

    //if (args.bounds.y - abs(position.y) < 0 &&
    //    position.y * velocity.y > 0)
    //{
    //    //velocity.y = -velocity.y;
    //    position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    //}

    velocity = scale(velocity, 1 - (args.friction * args.deltaTime));

    args.particles[index] = make_float4(position.x, position.y,
                                        velocity.x, velocity.y);
}

__global__ void tickNBody(KernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 particle = args.particles[index];
    float2 thisPosition = make_float2(particle.x, particle.y);

    float2 force = calculateForceMouse(particle.x, particle.y, args.particleMass,
                                  args.mousePos.x, args.mousePos.y, args.mouseMass);


    // Calculate particle forces
  /*  __shared__ float4 particles[BLOCK_SIZE];
    for (int particleI = 0; particleI < args.count; particleI += BLOCK_SIZE)
    {
        particles[threadIdx.x] = args.particles[particleI + threadIdx.x];
        __syncthreads();

        for (int subI = 0; subI < BLOCK_SIZE; subI++)
        {
            force = add(force, calculateForce(
                particle.x, particle.y, args.particleMass,
                particles[subI].x, particles[subI].y, args.particleMass)
                );
        }
    }*/

    float2 diff;
    float distance;
    float magnitude;
    for (int particleI = 0; particleI < args.count; particleI ++)
    {
    if (particleI != index)
    {
    diff.x = args.particles[particleI].x - particle.x;
    diff.y = args.particles[particleI].y - particle.y;

    distance = length(diff) + MIN_CALC_DISTANCE;
    magnitude = (GRAV_CONST * args.particleMass * args.particleMass) / (distance * distance);
    force = add(force, scale(normalize(diff), magnitude));
    }
    }


    // Calc resulting velocity
    float2 velocity = make_float2(particle.z, particle.w);
    velocity = add(velocity, scale(force, args.deltaTime)); // velocty += force * deltaTime

    float2 position = make_float2(particle.x, particle.y);
    position = add(position, scale(velocity, args.deltaTime));

    /*if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
        {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
        }

        if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
        {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
        }
        */
    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        //velocity.x = -velocity.x;
        position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        //velocity.y = -velocity.y;
        position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    }

    velocity = scale(velocity, 1 - args.friction * args.deltaTime);

    args.particles[index] = make_float4(position.x, position.y,
                                        velocity.x, velocity.y);
}


void launchCudaTick(float deltaTime, float particleMass, float mouseX, float mouseY, float mouseMass,
struct hipGraphicsResource* particleResource, bool mouseOnly, int particleCount, float2 bounds, float friction)
{
    size_t numBytes;
    KernelArgs args;
    float4* particles;
    hipError_t result = hipGraphicsMapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed mapping resource\n");
        exit(0);
    }

    result = hipGraphicsResourceGetMappedPointer((void**) &particles, &numBytes, particleResource);
    if (result != hipSuccess)
    {
        printf("failed grabbing resource\n");
        exit(0);
    }

    args.particles = particles;
    args.count = particleCount;
    args.deltaTime = deltaTime;
    args.particleMass = particleMass;
    args.mousePos = make_float2(mouseX, mouseY);
    args.mouseMass = mouseMass;
    args.friction = friction;
    args.bounds = bounds;

    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(particleCount / BLOCK_SIZE, 1, 1);

    if (mouseOnly)
    {
        tickMouseOnly<<<gridDim, blockDim>>>(args);
    }
    else
    {
        tickNBody<<<gridDim, blockDim>>>(args);
    }

    hipDeviceSynchronize();

    result = hipGraphicsUnmapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed unmapping resource\n");
        exit(0);
    }
}
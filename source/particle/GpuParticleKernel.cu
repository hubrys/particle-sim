#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.h"
#include <iostream>

#define BLOCK_SIZE 512
#define MIN_CALC_DISTANCE .01f
#define GRAV_CONST 1

typedef struct
{
    float4* particles;
    int count;
    float deltaTime;
    float particleMass;
    float2 mousePos;
    float mouseMass;
    float friction;
    float2 bounds;
} KernelArgs;

__global__ void tickMouseOnly(KernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 particle = args.particles[index];

    // calculate position diff
    float2 diff;
    diff.x = args.mousePos.x - particle.x;
    diff.y = args.mousePos.y - particle.y;

    float distance = length(diff) + MIN_CALC_DISTANCE;
    float magnitude = GRAV_CONST * args.mouseMass * args.particleMass / (distance);
    float2 velocity = scale(normalize(diff), magnitude * args.deltaTime);
    velocity.x += particle.z;
    velocity.y += particle.w;
    float2 position = make_float2(particle.x, particle.y);

    position.x += velocity.x * args.deltaTime;
    position.y += velocity.y * args.deltaTime;
    
    if (args.bounds.x - abs(position.x) < 0 &&
    position.x * velocity.x > 0)
    {
    velocity.x = -velocity.x;
    position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
    position.y * velocity.y > 0)
    {
    velocity.y = -velocity.y;
    position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
    }
    

    //if (args.bounds.x - abs(position.x) < 0 &&
    //    position.x * velocity.x > 0)
    //{
    //    //velocity.x = -velocity.x;
    //    position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    //}

    //if (args.bounds.y - abs(position.y) < 0 &&
    //    position.y * velocity.y > 0)
    //{
    //    //velocity.y = -velocity.y;
    //    position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    //}
    
    velocity = scale(velocity, 1 - args.friction * args.deltaTime);

    args.particles[index] = make_float4(position.x, position.y,
                                        velocity.x, velocity.y);
}

__global__ void tickNBody(KernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 particle = args.particles[index];
    float2 thisPosition = make_float2(particle.x, particle.y);

    // calculate position diff
    float2 diff = subtract(args.mousePos, thisPosition);

    // Calculate mouse force/velocity
    float distance = length(diff) + MIN_CALC_DISTANCE;
    float magnitude = GRAV_CONST * args.mouseMass * args.particleMass / distance;
    float2 force = scale(normalize(diff), magnitude);
    

    // Calculate particle forces
    __shared__ float4 particles[BLOCK_SIZE];
    for (int particleI = 0; particleI < args.count; particleI+=BLOCK_SIZE)
    {
        particles[threadIdx.x] = args.particles[particleI + threadIdx.x];
        __syncthreads();

        for (int subI = 0; subI < BLOCK_SIZE; subI++)
        {
            diff.x = particles[subI].x - particle.x;
            diff.y = particles[subI].y - particle.y;

            distance = length(diff);
            if (distance != 0)
            {
                distance += MIN_CALC_DISTANCE;
                magnitude = (GRAV_CONST * args.particleMass * args.particleMass) / (distance * distance);
                force = add(force, scale(normalize(diff), magnitude));
            }
        }
    }
    
    /*for (int particleI = 0; particleI < args.count; particleI ++)
    {
        if (particleI != index)
        {
            diff.x = args.particles[particleI].x - particle.x;
            diff.y = args.particles[particleI].y - particle.y;

            distance = length(diff) + MIN_CALC_DISTANCE;
            magnitude = (GRAV_CONST * args.particleMass * args.particleMass) / (distance * distance);
            force = add(force, scale(normalize(diff), magnitude));
        }
    }*/


    // Calc resulting velocity
    float2 velocity = make_float2(particle.z, particle.w);
    velocity = add(velocity, scale(force, args.deltaTime)); // velocty += force * deltaTime

    float2 position = make_float2(particle.x, particle.y);
    position = add(position, scale(velocity, args.deltaTime));

    /*if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
    }
*/
    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        //velocity.x = -velocity.x;
        position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        //velocity.y = -velocity.y;
        position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    }

    velocity = scale(velocity, 1 - args.friction * args.deltaTime);

    args.particles[index] = make_float4(position.x, position.y,
                                        velocity.x, velocity.y);
}


void launchCudaTick(float deltaTime, float particleMass, float mouseX, float mouseY, float mouseMass,
struct hipGraphicsResource* particleResource, bool mouseOnly, int particleCount, float2 bounds, float friction)
{
    size_t numBytes;
    KernelArgs args;
    float4* particles;
    hipError_t result = hipGraphicsMapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed mapping resource\n");
        exit(0);
    }

    result = hipGraphicsResourceGetMappedPointer((void**)&particles, &numBytes, particleResource);
    if (result != hipSuccess)
    {
        printf("failed grabbing resource\n");
        exit(0);
    }

    args.particles = particles;
    args.count = particleCount;
    args.deltaTime = deltaTime;
    args.particleMass = particleMass;
    args.mousePos = make_float2(mouseX, mouseY);
    args.mouseMass = mouseMass;
    args.friction = friction;
    args.bounds = bounds;

    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(particleCount / BLOCK_SIZE, 1, 1);

    if (mouseOnly)
    {
        tickMouseOnly<<<gridDim, blockDim>>>(args);
    } 
    else
    {
        tickNBody<<<gridDim, blockDim>>>(args);
    }

    hipDeviceSynchronize();

    result = hipGraphicsUnmapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed unmapping resource\n");
        exit(0);
    }
}
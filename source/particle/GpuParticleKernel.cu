#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.h"
#include <iostream>

typedef struct
{
    float4* particles;
    int count;
    float deltaTime;
    float particleMass;
    float2 mousePos;
    float mouseMass;
    float friction;
    float2 bounds;
} KernelArgs;

__global__ void tickMouseOnly(KernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 particle = args.particles[index];

    float2 velocity = scale(
        calculateForceMouse(particle.x, particle.y, args.particleMass, args.mousePos.x, args.mousePos.y, args.mouseMass),
        args.deltaTime
        );

    velocity.x += particle.z;
    velocity.y += particle.w;
    float2 position = make_float2(particle.x, particle.y);

    position.x += velocity.x * args.deltaTime;
    position.y += velocity.y * args.deltaTime;

    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
    }


    //if (args.bounds.x - abs(position.x) < 0 &&
    //    position.x * velocity.x > 0)
    //{
    //    //velocity.x = -velocity.x;
    //    position.x = position.x > 0 ? -args.bounds.x : args.bounds.x;
    //}

    //if (args.bounds.y - abs(position.y) < 0 &&
    //    position.y * velocity.y > 0)
    //{
    //    //velocity.y = -velocity.y;
    //    position.y = position.y > 0 ? -args.bounds.y : args.bounds.y;
    //}

    velocity = scale(velocity, 1 - (args.friction * args.deltaTime));

    args.particles[index] = make_float4(position.x, position.y,
                                        velocity.x, velocity.y);
}

__global__ void tickNBody(KernelArgs args)
{
    int index = PARTICLES_PER_THREAD * blockIdx.x * blockDim.x + threadIdx.x;

    float4 particles[PARTICLES_PER_THREAD];
    particles[0] = args.particles[index];
    particles[1] = args.particles[index + BLOCK_SIZE];
    particles[2] = args.particles[index + 2 * BLOCK_SIZE];
    particles[3] = args.particles[index + 3 * BLOCK_SIZE];

    float2 forces[PARTICLES_PER_THREAD];
    forces[0] = calculateForceMouse(particles[0].x, particles[0].y, args.particleMass,
                                  args.mousePos.x, args.mousePos.y, args.mouseMass);
    forces[1] = calculateForceMouse(particles[1].x, particles[1].y, args.particleMass,
                                    args.mousePos.x, args.mousePos.y, args.mouseMass);
    forces[2] = calculateForceMouse(particles[2].x, particles[2].y, args.particleMass,
                                    args.mousePos.x, args.mousePos.y, args.mouseMass);
    forces[3] = calculateForceMouse(particles[3].x, particles[3].y, args.particleMass,
                                    args.mousePos.x, args.mousePos.y, args.mouseMass);


    // Calculate particle forces
    __shared__ float4 otherParticles[BLOCK_SIZE];
    for (int particleI = 0; particleI < args.count; particleI += BLOCK_SIZE)
    {
        otherParticles[threadIdx.x] = args.particles[particleI + threadIdx.x];
        __syncthreads();

        for (int subI = 0; subI < BLOCK_SIZE; subI++)
        {
            forces[0] = add(forces[0], calculateForce(
                particles[0].x, particles[0].y, args.particleMass,
                otherParticles[subI].x, otherParticles[subI].y, args.particleMass)
                );
            forces[1] = add(forces[1], calculateForce(
                particles[1].x, particles[1].y, args.particleMass,
                otherParticles[subI].x, otherParticles[subI].y, args.particleMass)
                );
            forces[2] = add(forces[2], calculateForce(
                particles[2].x, particles[2].y, args.particleMass,
                otherParticles[subI].x, otherParticles[subI].y, args.particleMass)
                );
            forces[3] = add(forces[3], calculateForce(
                particles[3].x, particles[3].y, args.particleMass,
                otherParticles[subI].x, otherParticles[subI].y, args.particleMass)
                );
        }
    }

    // Calc resulting velocity
    float2 velocities[PARTICLES_PER_THREAD];
    velocities[0] = add(make_float2(particles[0].z, particles[0].w), 
                    scale(forces[0], args.deltaTime)); // velocty += force * deltaTime
    velocities[1] = add(make_float2(particles[1].z, particles[1].w),
                        scale(forces[1], args.deltaTime)); // velocty += force * deltaTime
    velocities[2] = add(make_float2(particles[2].z, particles[2].w),
                        scale(forces[2], args.deltaTime)); // velocty += force * deltaTime
    velocities[3] = add(make_float2(particles[3].z, particles[3].w),
                        scale(forces[3], args.deltaTime)); // velocty += force * deltaTime

    float2 positions[PARTICLES_PER_THREAD];
    positions[0] = add(make_float2(particles[0].x, particles[0].y),
                       scale(velocities[0], args.deltaTime));
    positions[1] = add(make_float2(particles[1].x, particles[1].y),
                       scale(velocities[1], args.deltaTime));
    positions[2] = add(make_float2(particles[2].x, particles[2].y),
                       scale(velocities[2], args.deltaTime));
    positions[3] = add(make_float2(particles[3].x, particles[3].y),
                       scale(velocities[3], args.deltaTime));

    // bounds checking
    if (args.bounds.x - abs(positions[0].x) < 0 &&
        positions[0].x * velocities[1].x > 0)
    {
        //velocity.x = -velocity.x;
        positions[0].x = positions[0].x > 0 ? -args.bounds.x : args.bounds.x;
    }
    if (args.bounds.y - abs(positions[0].y) < 0 &&
        positions[0].y * velocities[1].y > 0)
    {
        //velocity.y = -velocity.y;
        positions[0].y = positions[0].y > 0 ? -args.bounds.y : args.bounds.y;
    }

    if (args.bounds.x - abs(positions[1].x) < 0 &&
        positions[1].x * velocities[1].x > 0)
    {
        //velocity.x = -velocity.x;
        positions[1].x = positions[1].x > 0 ? -args.bounds.x : args.bounds.x;
    }
    if (args.bounds.y - abs(positions[1].y) < 0 &&
        positions[1].y * velocities[1].y > 0)
    {
        //velocity.y = -velocity.y;
        positions[1].y = positions[1].y > 0 ? -args.bounds.y : args.bounds.y;
    }

    if (args.bounds.x - abs(positions[2].x) < 0 &&
        positions[2].x * velocities[2].x > 0)
    {
        //velocity.x = -velocity.x;
        positions[2].x = positions[2].x > 0 ? -args.bounds.x : args.bounds.x;
    }
    if (args.bounds.y - abs(positions[2].y) < 0 &&
        positions[2].y * velocities[2].y > 0)
    {
        //velocity.y = -velocity.y;
        positions[2].y = positions[2].y > 0 ? -args.bounds.y : args.bounds.y;
    }

    if (args.bounds.x - abs(positions[3].x) < 0 &&
        positions[3].x * velocities[3].x > 0)
    {
        //velocity.x = -velocity.x;
        positions[3].x = positions[3].x > 0 ? -args.bounds.x : args.bounds.x;
    }
    if (args.bounds.y - abs(positions[3].y) < 0 &&
        positions[3].y * velocities[3].y > 0)
    {
        //velocity.y = -velocity.y;
        positions[3].y = positions[3].y > 0 ? -args.bounds.y : args.bounds.y;
    }

    // Apply friction
    velocities[0] = scale(velocities[0], 1 - args.friction * args.deltaTime);
    velocities[1] = scale(velocities[1], 1 - args.friction * args.deltaTime);
    velocities[2] = scale(velocities[2], 1 - args.friction * args.deltaTime);
    velocities[3] = scale(velocities[3], 1 - args.friction * args.deltaTime);

    args.particles[index] = make_float4(positions[0].x, positions[0].y,
                                        velocities[0].x, velocities[0].y);
    args.particles[index + BLOCK_SIZE] = make_float4(positions[1].x, positions[1].y,
                                        velocities[1].x, velocities[1].y);
    args.particles[index + 2 * BLOCK_SIZE] = make_float4(positions[2].x, positions[2].y,
                                                     velocities[2].x, velocities[2].y);
    args.particles[index + 3 * BLOCK_SIZE] = make_float4(positions[3].x, positions[3].y,
                                                     velocities[3].x, velocities[3].y);
}


void launchCudaTick(float deltaTime, float particleMass, float mouseX, float mouseY, float mouseMass,
struct hipGraphicsResource* particleResource, bool mouseOnly, int particleCount, float2 bounds, float friction)
{
    size_t numBytes;
    KernelArgs args;
    float4* particles;
    hipError_t result = hipGraphicsMapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed mapping resource\n");
        exit(0);
    }

    result = hipGraphicsResourceGetMappedPointer((void**) &particles, &numBytes, particleResource);
    if (result != hipSuccess)
    {
        printf("failed grabbing resource\n");
        exit(0);
    }

    args.particles = particles;
    args.count = particleCount;
    args.deltaTime = deltaTime;
    args.particleMass = particleMass;
    args.mousePos = make_float2(mouseX, mouseY);
    args.mouseMass = mouseMass;
    args.friction = friction;
    args.bounds = bounds;

    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(particleCount / (BLOCK_SIZE * PARTICLES_PER_THREAD), 1, 1);

    if (mouseOnly)
    {
        tickMouseOnly<<<gridDim, blockDim>>>(args);
    }
    else
    {
        tickNBody<<<gridDim, blockDim>>>(args);
    }

    hipDeviceSynchronize();

    result = hipGraphicsUnmapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed unmapping resource\n");
        exit(0);
    }
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.h"
#include <iostream>

#define BLOCK_SIZE 256
#define MIN_CALC_DISTANCE .01f
#define GRAV_CONST 1

typedef struct
{
    float4* particles;
    int count;
    float deltaTime;
    float particleMass;
    float2 mousePos;
    float mouseMass;
    float friction;
    float2 bounds;
} KernelArgs;

__global__ void kernelTick(KernelArgs args)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 particle = args.particles[index];

    // calculate position diff
    float2 diff;
    diff.x = args.mousePos.x - particle.x;
    diff.y = args.mousePos.y - particle.y;

    float distance = length(diff) + MIN_CALC_DISTANCE;
    float magnitude = GRAV_CONST * args.mouseMass * args.particleMass / distance;
    float2 velocity = scale(normalize(diff), magnitude * args.deltaTime);
    velocity.x += particle.z;
    velocity.y += particle.w;
    float2 position = make_float2(particle.x, particle.y);

    position.x += velocity.x * args.deltaTime;
    position.y += velocity.y * args.deltaTime;
    
    if (args.bounds.x - abs(position.x) < 0 &&
        position.x * velocity.x > 0)
    {
        velocity.x = -velocity.x;
        position.x = position.x < 0 ? -args.bounds.x : args.bounds.x;
    }

    if (args.bounds.y - abs(position.y) < 0 &&
        position.y * velocity.y > 0)
    {
        velocity.y = -velocity.y;
        position.y = position.y < 0 ? -args.bounds.y : args.bounds.y;
    }
    
    velocity = scale(velocity, 1 - args.friction * args.deltaTime);

    args.particles[index] = make_float4(position.x, position.y,
                                        velocity.x, velocity.y);
}


void launchCudaTick(float deltaTime, float particleMass, float mouseX, float mouseY, float mouseMass,
struct hipGraphicsResource* particleResource, bool mouseOnly, int particleCount, float2 bounds)
{
    size_t numBytes;
    KernelArgs args;
    float4* particles;
    hipError_t result = hipGraphicsMapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed mapping resource\n");
        exit(0);
    }

    result = hipGraphicsResourceGetMappedPointer((void**)&particles, &numBytes, particleResource);
    if (result != hipSuccess)
    {
        printf("failed grabbing resource\n");
        exit(0);
    }

    args.particles = particles;
    args.count = particleCount;
    args.deltaTime = deltaTime;
    args.particleMass = particleMass;
    args.mousePos = make_float2(mouseX, mouseY);
    args.mouseMass = mouseMass;
    args.friction = .9f;
    args.bounds = bounds;

    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(particleCount / BLOCK_SIZE, 1, 1);

    kernelTick<<<gridDim, blockDim>>>(args);

    hipDeviceSynchronize();

    result = hipGraphicsUnmapResources(1, &particleResource, 0);
    if (result != hipSuccess)
    {
        printf("failed unmapping resource\n");
        exit(0);
    }
}